#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <freshman.h>
#include <>
#include <math.h>
#define N 3000000

__global__ void kernel_1() {
	double sum = 0.0;
	for (int i = 0; i < N; i++)
		sum = sum + tan(0.1) * tan(0.1);
}

__global__ void kernel_2() {
	double sum = 0.0;
	for (int i = 0; i < N; i++)
		sum = sum + tan(0.1) * tan(0.1);
}

__global__ void kernel_3() {
	double sum = 0.0;
	for (int i = 0; i < N; i++)
		sum = sum + tan(0.1) * tan(0.1);
}

__global__ void kernel_4() {
	double sum = 0.0;
	for (int i = 0; i < N; i++)
		sum = sum + tan(0.1) * tan(0.1);
}

int main() {

	/*setenv("CUDA_DEVICE_MAX_CONNECTIONS", "32", 1);*/  // windows����û��setenv, ��������Linuxϵͳ
	putenv("CUDA_DEVICE_MAX_CONNECTIONS");

	int dev = 0;
	hipSetDevice(dev);
	int n_stream = 16;
	hipStream_t* stream = (hipStream_t*)malloc(n_stream * sizeof(hipStream_t));

	for (int i = 0; i < n_stream; i++) {
		hipStreamCreate(&stream[i]);
	}

	dim3 block(1);
	dim3 grid(1);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	for (int i = 0; i < n_stream; i++) {
		kernel_1 << <grid, block, 0, stream[i] >> > ();
		kernel_2 << <grid, block, 0, stream[i] >> > ();
		kernel_3 << <grid, block, 0, stream[i] >> > ();
		kernel_4 << <grid, block, 0, stream[i] >> > ();
	}

	hipEventRecord(stop, 0);
	CHECK(hipEventSynchronize(stop));

	float elapsed_time;
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("elapsed time: %f ms \n", elapsed_time);

	for (int i = 0; i < n_stream; i++) {
		hipStreamDestroy(stream[i]);
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(stream);
	CHECK(hipDeviceReset());
	return 0;
}