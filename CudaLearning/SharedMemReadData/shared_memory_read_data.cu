#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <freshman.h>
#include <>

#define BDIMX 32
#define BDIMY 32

#define BDIMX_RECT 32
#define BDIMY_RECT 16
#define IPAD 1


__global__ void warmup(int* out) {
	__shared__ int tile[BDIMY][BDIMX];
	unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

	tile[threadIdx.y][threadIdx.x] = idx;
	__syncthreads();
	out[idx] = tile[threadIdx.y][threadIdx.x];
}

__global__ void setRowReadRow(int* out) {
	__shared__ int tile[BDIMY][BDIMX];
	unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

	tile[threadIdx.y][threadIdx.x] = idx;
	__syncthreads();
	out[idx] = tile[threadIdx.y][threadIdx.x];
}

__global__ void setColReadCol(int* out) {
	__shared__ int tile[BDIMY][BDIMX];
	unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

	tile[threadIdx.x][threadIdx.y] = idx;
	__syncthreads();
	out[idx] = tile[threadIdx.x][threadIdx.y];
}

__global__ void setColReadRow(int* out) {
	__shared__ int tile[BDIMY][BDIMX];
	unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

	tile[threadIdx.x][threadIdx.y] = idx;
	__syncthreads();
	out[idx] = tile[threadIdx.y][threadIdx.x];
}

__global__ void setRowReadCol(int* out) {
	__shared__ int tile[BDIMY][BDIMX];
	unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;

	tile[threadIdx.y][threadIdx.x] = idx;
	__syncthreads();
	out[idx] = tile[threadIdx.x][threadIdx.y];
}

__global__ void setRowReadColDyn(int* out) {
	extern __shared__ int tile[];
	unsigned int row_idx = threadIdx.y * blockDim.x + threadIdx.x;
	unsigned int col_idx = threadIdx.x * blockDim.y + threadIdx.y;
	tile[row_idx] = row_idx;
	__syncthreads();
	out[row_idx] = tile[col_idx];
}

__global__ void setRowReadColIpad(int* out) {
	__shared__ int tile[BDIMY][BDIMX + IPAD];
	
}